// Compile: nvcc -arch=sm_61 -std=c++11 assignment5-p3.cu -o assignment5-p3


#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdint>
#include <iostream>
#include <sys/time.h>

#define SIZE 128
#define BLOCK_SIZE 16
#define THRESHOLD (0.000001)

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) {
    cout << "Error return from gettimeofday: " << stat << "\n";
  }
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

__host__ void ATAonCPU(double* M, double* P) {
  for (int k = 0; k < SIZE; k++) {
    for (int i = 0; i < SIZE; i++) {
      for (int j = 0; j < SIZE; j++)
        P[i*SIZE + j] += M[k*SIZE + i] * M[k*SIZE + j];
    }
  }
}

__host__ void check_result(double* Test, double* Ref) {
  double maxdiff = 0, rel_diff = 0;
  int numdiffs = 0;

  for (int i = 0; i < SIZE; i++) {
    for (int j = 0; j < SIZE; j++) {
      rel_diff = (Test[i*SIZE + j] - Ref[i*SIZE + j]);
      if (fabs(rel_diff) > THRESHOLD) {
        numdiffs++;
        if (rel_diff > maxdiff)
          maxdiff = rel_diff;
      }
    }
  }
  if (numdiffs > 0)
    cout << numdiffs << " Diffs found over THRESHOLD " << THRESHOLD << " Max Diff = " << maxdiff
         << "\n";
  else
    cout << "No differences found between base and test versions\n";
}

// SB: Implement your kernel here
__global__ void ATAkernel(double* M, double* P) {

  int i =  blockIdx.y*blockDim.y + threadIdx.y;
  int j =  blockIdx.x*blockDim.x + threadIdx.x;
  
  if(i < SIZE and j < SIZE) {
    for (int k = 0; k < SIZE; k++)
      P[i*SIZE + j] += M[k*SIZE + i] * M[k*SIZE + j];
  }
}

int main() {
  cout << "Matrix Size = " << SIZE << "\n";

  double* A = new double[SIZE*SIZE];

  double* O_s = new double[SIZE*SIZE];

  double* O_p = new double[SIZE*SIZE];

  for (int i = 0; i < SIZE; i++) {
    for (int j = 0; j < SIZE; j++) {
      A[i*SIZE + j] = i * j * 0.25;
      O_s[i*SIZE + j] = 0;
      O_p[i*SIZE + j] = 0;
    }
  }

  double clkbegin, clkend;
  double t;

  clkbegin = rtclock();
  ATAonCPU(A, O_s);
  clkend = rtclock();
  t = clkend - clkbegin;
  cout << "A^T.A on CPU: " << (2.0 * SIZE * SIZE * SIZE / t / 1.0e9)
       << " GFLOPS; Time = " << t * 1000 << " msec\n";

  hipEvent_t start, end;

  gpuErrchk( hipEventCreate(&start) );
  gpuErrchk( hipEventCreate(&end) );
  
  // SB: Write your GPU kernel here
  double *O_p_c, *A_c;
  gpuErrchk( hipMalloc((void**)&O_p_c, SIZE*SIZE*sizeof(double)) );
  gpuErrchk( hipMalloc((void**)&A_c, SIZE*SIZE*sizeof(double)) );
  gpuErrchk( hipMemcpy(O_p_c, O_p, SIZE*SIZE*sizeof(double), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(A_c, A, SIZE*SIZE*sizeof(double), hipMemcpyHostToDevice) );
  dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridSize((SIZE + blockSize.x - 1)/blockSize.x, (SIZE + blockSize.y - 1)/blockSize.y);
  
  gpuErrchk( hipEventRecord(start, 0) );
  ATAkernel<<<gridSize, blockSize>>>(A_c, O_p_c);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipEventRecord(end, 0) );

  gpuErrchk( hipMemcpy(O_p, O_p_c, SIZE*SIZE*sizeof(double), hipMemcpyDeviceToHost) );
  
  gpuErrchk( hipDeviceSynchronize() );

  float kernel_time = 0;
  gpuErrchk( hipEventElapsedTime(&kernel_time, start, end) );

  cout << "A^T.A on GPU: " << (2.0 * SIZE * SIZE * SIZE / t / 1.0e9)
       << " GFLOPS; Time = " << kernel_time << " msec\n";

  check_result(O_p, O_s);

  gpuErrchk( hipFree(O_p_c) );
  gpuErrchk( hipFree(A_c) );
  
  free(O_s);
  free(O_p);
  free(A);

  return EXIT_SUCCESS;
}
