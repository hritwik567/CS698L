// Compile: nvcc -arch=sm_61 -std=c++11 assignment5-p3.cu -o assignment5-p3


#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdint>
#include <iostream>
#include <sys/time.h>

#define SIZE 1024
#define BLOCK_SIZE 16
#define THRESHOLD (0.000001)

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) {
    cout << "Error return from gettimeofday: " << stat << "\n";
  }
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

__host__ void ATAonCPU(double* M, double* P) {
  for (int k = 0; k < SIZE; k++) {
    for (int i = 0; i < SIZE; i++) {
      for (int j = 0; j < SIZE; j++)
        P[i*SIZE + j] += M[k*SIZE + i] * M[k*SIZE + j];
    }
  }
}

__host__ void check_result(double* Test, double* Ref) {
  double maxdiff = 0, rel_diff = 0;
  int numdiffs = 0;

  for (int i = 0; i < SIZE; i++) {
    for (int j = 0; j < SIZE; j++) {
      rel_diff = (Test[i*SIZE + j] - Ref[i*SIZE + j]);
      if (fabs(rel_diff) > THRESHOLD) {
        printf("%f %f %f\n",Test[i*SIZE + j], Ref[i*SIZE + j], rel_diff);
        numdiffs++;
        if (rel_diff > maxdiff)
          maxdiff = rel_diff;
      }
    }
  }
  if (numdiffs > 0)
    cout << numdiffs << " Diffs found over THRESHOLD " << THRESHOLD << " Max Diff = " << maxdiff
         << "\n";
  else
    cout << "No differences found between base and test versions\n";
}

// SB: Implement your kernel here
__global__ void ATAkernel(double* M, double* P) {

  if(blockIdx.x < blockIdx.y) return;
  double sum = 0;

  uint64_t i = blockIdx.y*blockDim.y + threadIdx.y;
  uint64_t j = blockIdx.x*blockDim.x + threadIdx.x;

  __shared__ double A_t[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double B_t[BLOCK_SIZE][BLOCK_SIZE];

  for (uint64_t tid = 0; tid < SIZE/blockDim.x; tid++) {
    A_t[threadIdx.y][threadIdx.x] = M[(tid * blockDim.x + threadIdx.x) * SIZE + i];
    B_t[threadIdx.y][threadIdx.x] = M[(tid * blockDim.y + threadIdx.y) * SIZE + j];
   
    __syncthreads();

    sum += A_t[threadIdx.y][0] * B_t[0][threadIdx.x]
          + A_t[threadIdx.y][1] * B_t[1][threadIdx.x]
          + A_t[threadIdx.y][2] * B_t[2][threadIdx.x]
          + A_t[threadIdx.y][3] * B_t[3][threadIdx.x]
          + A_t[threadIdx.y][4] * B_t[4][threadIdx.x]
          + A_t[threadIdx.y][5] * B_t[5][threadIdx.x]
          + A_t[threadIdx.y][6] * B_t[6][threadIdx.x]
          + A_t[threadIdx.y][7] * B_t[7][threadIdx.x]
          + A_t[threadIdx.y][8] * B_t[8][threadIdx.x]
          + A_t[threadIdx.y][9] * B_t[9][threadIdx.x]
          + A_t[threadIdx.y][10] * B_t[10][threadIdx.x]
          + A_t[threadIdx.y][11] * B_t[11][threadIdx.x]
          + A_t[threadIdx.y][12] * B_t[12][threadIdx.x]
          + A_t[threadIdx.y][13] * B_t[13][threadIdx.x]
          + A_t[threadIdx.y][14] * B_t[14][threadIdx.x]
          + A_t[threadIdx.y][15] * B_t[15][threadIdx.x];

    __syncthreads();
  }
  
  P[i * SIZE + j] = sum;
  if(blockIdx.x > blockIdx.y) P[j * SIZE + i] = sum;
}

int main() {
  cout << "Matrix Size = " << SIZE << "\n";

  double* A = new double[SIZE*SIZE];

  double* O_s = new double[SIZE*SIZE];

  double* O_p = new double[SIZE*SIZE];

  for (int i = 0; i < SIZE; i++) {
    for (int j = 0; j < SIZE; j++) {
      // A[i*SIZE + j] = random() * 0.25;
      A[i*SIZE + j] = i * (j-i) * 0.25;
      O_s[i*SIZE + j] = 0;
      O_p[i*SIZE + j] = 0;
    }
  }

  double clkbegin, clkend;
  double t;

  clkbegin = rtclock();
  ATAonCPU(A, O_s);
  clkend = rtclock();
  t = clkend - clkbegin;
  cout << "A^T.A on CPU: " << (2.0 * SIZE * SIZE * SIZE / t / 1.0e9)
       << " GFLOPS; Time = " << t * 1000 << " msec\n";

  hipEvent_t start, end;

  gpuErrchk( hipEventCreate(&start) );
  gpuErrchk( hipEventCreate(&end) );
  
  // SB: Write your GPU kernel here
  double *O_p_c, *A_c;
  gpuErrchk( hipMalloc((void**)&O_p_c, SIZE*SIZE*sizeof(double)) );
  gpuErrchk( hipMalloc((void**)&A_c, SIZE*SIZE*sizeof(double)) );
  dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
  dim3 gridSize((SIZE + blockSize.x - 1)/blockSize.x, (SIZE + blockSize.y - 1)/blockSize.y);
  
  gpuErrchk( hipEventRecord(start, 0) );
  gpuErrchk( hipMemcpy(O_p_c, O_p, SIZE*SIZE*sizeof(double), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(A_c, A, SIZE*SIZE*sizeof(double), hipMemcpyHostToDevice) );
  ATAkernel<<<gridSize, blockSize>>>(A_c, O_p_c);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipMemcpy(O_p, O_p_c, SIZE*SIZE*sizeof(double), hipMemcpyDeviceToHost) );
  gpuErrchk( hipEventRecord(end, 0) );

  
  gpuErrchk( hipDeviceSynchronize() );

  float kernel_time = 0;
  gpuErrchk( hipEventElapsedTime(&kernel_time, start, end) );

  cout << "A^T.A on GPU: " << (2.0 * SIZE * SIZE * SIZE / t / 1.0e9)
       << " GFLOPS; Time = " << kernel_time << " msec\n";

  check_result(O_p, O_s);

  gpuErrchk( hipFree(O_p_c) );
  gpuErrchk( hipFree(A_c) );
  
  free(O_s);
  free(O_p);
  free(A);

  return EXIT_SUCCESS;
}
