// Compile: nvcc -arch=sm_61 -std=c++11 assignment5-p2.cu -o assignment5-p2


#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdint>
#include <iostream>
#include <sys/time.h>

#define THRESHOLD (0.000001)

#define SIZE1 4096
#define SIZE2 4097
#define ITER 100

using namespace std;

__global__ void kernel1(double** A) {
  // SB: Write the first kernel here
  int j = threadIdx.x;
  for (int k = 0; k < ITER; k++) {
    for (int i = 1; i < SIZE1; i++) {
      A[i][j + 1] = A[i - 1][j + 1] + A[i][j + 1];
    }
  }
}

__global__ void kernel2(double** A) {
  // SB: Write the second kernel here
  int j = threadIdx.x;
  for (int k = 0; k < ITER; k++) {
    for (int i = 1; i < SIZE1; i++) {
      A[i][j + 1] = A[i - 1][j + 1] + A[i][j + 1];
    }
  }
}

__host__ void serial(double** A) {
  for (int k = 0; k < ITER; k++) {
    for (int i = 1; i < SIZE1; i++) {
      for (int j = 0; j < SIZE1 - 1; j++) {
        A[i][j + 1] = A[i - 1][j + 1] + A[i][j + 1];
      }
    }
  }
}

__host__ void check_result(double** w_ref, double** w_opt, uint64_t size) {
  double maxdiff = 0.0, this_diff = 0.0;
  int numdiffs = 0;

  for (uint64_t i = 0; i < size; i++) {
    for (uint64_t j = 0; j < size; j++) {
      this_diff = w_ref[i][j] - w_opt[i][j];
      if (fabs(this_diff) > THRESHOLD) {
        numdiffs++;
        if (this_diff > maxdiff)
          maxdiff = this_diff;
      }
    }
  }

  if (numdiffs > 0) {
    cout << numdiffs << " Diffs found over THRESHOLD " << THRESHOLD << "; Max Diff = " << maxdiff
         << endl;
  } else {
    cout << "No differences found between base and test versions\n";
  }
}

__host__ double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) {
    cout << "Error return from gettimeofday: " << stat << "\n";
  }
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

int main() {
  double** A_ser = new double*[SIZE1];
  double** A_k1 = new double*[SIZE1];
  for (int i = 0; i < SIZE1; i++) {
    A_ser[i] = new double[SIZE1];
    A_k1[i] = new double[SIZE1];
  }

  double** A_k2 = new double*[SIZE2];
  for (int i = 0; i < SIZE2; i++) {
    A_k2[i] = new double[SIZE2];
  }

  for (int i = 0; i < SIZE1; i++) {
    for (int j = 0; i < SIZE1; j++) {
      A_ser[i][j] = i + j;
      A_k1[i][j] = i + j;
    }
  }
  for (int i = 0; i < SIZE2; i++) {
    for (int j = 0; i < SIZE2; j++) {
      A_k2[i][j] = i + j;
    }
  }

  double clkbegin, clkend;
  double t;

  clkbegin = rtclock();
  serial(A_ser);
  clkend = rtclock();
  t = clkend - clkbegin;
  cout << "Serial code on CPU: " << (1.0 * SIZE1 * SIZE1 * ITER / t / 1.0e9)
       << " GFLOPS; Time = " << t * 1000 << " msec" << endl;

  // cudaError_t status;
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);
  // SB: Write your first GPU kernel here
  kernel1<<<1, SIZE1 - 1>>>(A_k1);
  hipEventRecord(end, 0);
  float kernel_time;
  hipEventElapsedTime(&kernel_time, start, end);
  check_result(A_ser, A_k1, SIZE1);
  cout << "Kernel 1 on GPU: " << (1.0 * SIZE1 * SIZE1 * ITER / t / 1.0e9)
       << " GFLOPS; Time = " << kernel_time << " msec" << endl;

  hipEventRecord(start, 0);
  // SB: Write your second GPU kernel here
  kernel2<<<1, SIZE2 - 1>>>(A_k2);
  hipEventRecord(end, 0);
  hipEventElapsedTime(&kernel_time, start, end);
  check_result(A_ser, A_k2, SIZE2);
  cout << "Kernel 2 on GPU: " << (1.0 * SIZE2 * SIZE2 * ITER / t / 1.0e9)
       << " GFLOPS; Time = " << kernel_time << " msec" << endl;

  return EXIT_SUCCESS;
}
